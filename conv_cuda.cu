#include "hip/hip_runtime.h"
#include <iostream>

#include "convolution.h"
#include "array_management.h"
#include "test_cases.h"

using namespace std;

int main(int argc, char* argv[]) {

	if(argc == 1)
	{
		testing::InitGoogleTest(&argc, argv);
	  return RUN_ALL_TESTS();
	}
	else if (argc < 4) {
		cout << "Usage: srun out <int: dimension of input matrix> <int: dimension of kernel> <blocksize>\n";
		return 0;
	}

	uint32_t MATDIM = strtol(argv[1], NULL, 10);
	uint32_t KERDIM = strtol(argv[2], NULL, 10);
	uint32_t N = strtol(argv[3], NULL, 10);
	uint32_t grid_size = ceil((MATDIM-KERDIM+1) * (MATDIM-KERDIM+1) / ((double) N));

	uint32_t h_MATDIM[1];
	h_MATDIM[0] = MATDIM;
	uint32_t h_KERDIM[1];
	h_KERDIM[0] = KERDIM;

	struct timespec tstart={0,0}, tend={0,0};
	double elapsed;

	// Matrix (double)
	double* 	h_mat = new double[MATDIM*MATDIM];
	// Kernel (double)
	double* 	h_ker = new double[KERDIM*KERDIM];

    // Matrix (bits)
	unsigned char* 	h_mat_bin = new unsigned char[(uint32_t) ceil(MATDIM*MATDIM/8.0)];
    // Kernel (bits)
	unsigned char* 	h_ker_bin = new unsigned char[(uint32_t) ceil(KERDIM*KERDIM/8.0)];

	// Result of standard convolution
	double* 	h_res_standard = new double[(MATDIM-KERDIM+1)*(MATDIM-KERDIM+1)];
	// Result of convolution with binary weights
	double* 	h_res_binW = new double[(MATDIM-KERDIM+1)*(MATDIM-KERDIM+1)];

	double* 	new_h_res_binW = new double[(MATDIM-KERDIM+1)*(MATDIM-KERDIM+1)];
	// Result of convolution with binary weights and binary inputs
	unsigned char* 	h_res_binWbinI = new unsigned char[(MATDIM-KERDIM+1)*(MATDIM-KERDIM+1)];

	unsigned char* 	new_h_res_binWbinI = new unsigned char[(MATDIM-KERDIM+1)*(MATDIM-KERDIM+1)];

	uint32_t mat_size = 			MATDIM*MATDIM * sizeof(double);
	uint32_t ker_size = 			KERDIM*KERDIM * sizeof(double);
	uint32_t mat_bin_size = 		(uint32_t) ceil(MATDIM*MATDIM/8.0) * sizeof(unsigned char);
	uint32_t ker_bin_size = 		(uint32_t) ceil(KERDIM*KERDIM/8.0) * sizeof(unsigned char);
	uint32_t res_standard_size =	(MATDIM-KERDIM+1)*(MATDIM-KERDIM+1) * sizeof(double);
	uint32_t res_binW_size =		(MATDIM-KERDIM+1)*(MATDIM-KERDIM+1) * sizeof(double);
	uint32_t res_binWbinI_size =	(MATDIM-KERDIM+1)*(MATDIM-KERDIM+1) * sizeof(unsigned char);

	// Pointers for allocation on device
	uint32_t *d_MATDIM, *d_KERDIM;
	double *d_mat, *d_ker, *d_res_standard, *d_res_binW, *new_d_res_binW;
	unsigned char *d_mat_bin, *d_ker_bin, *d_res_binWbinI, *new_d_res_binWbinI;

	// Allocate all matrices on device (hipFree later!)
	hipMalloc((void**) &d_mat, mat_size);
	hipMalloc((void**) &d_ker, ker_size);
	hipMalloc((void**) &d_mat_bin, mat_bin_size);
	hipMalloc((void**) &d_ker_bin, ker_bin_size);
	hipMalloc((void**) &d_res_standard, res_standard_size);
	hipMalloc((void**) &d_res_binW, res_binW_size);
	hipMalloc((void**) &d_res_binWbinI, res_binWbinI_size);

	hipMalloc((void**) &new_d_res_binW, res_binW_size);
	hipMalloc((void**) &new_d_res_binWbinI, res_binWbinI_size);

	hipMalloc((void**) &d_MATDIM, sizeof(uint32_t));
	hipMalloc((void**) &d_KERDIM, sizeof(uint32_t));

	// Seed for random number generation
	srand(time(NULL));

	// Randomize the values of the double matrix with values -1.0 ... 1.0
	initMat(MATDIM, h_mat);
    // Convert the double matrix into binary (0 = -1, 1 = 1)
	convertToBinary(MATDIM, h_mat, (uint32_t) ceil(MATDIM*MATDIM/8.0), h_mat_bin);
	// TODO DEBUG: Print the binary matrix.
	if(argc == 4)
		printBinary(MATDIM, (uint32_t) ceil(MATDIM*MATDIM/8.0), h_mat_bin);

  //TODO DEBUG: Print the double matrix.
   printMatrix(MATDIM, h_mat);

	initMat(KERDIM, h_ker);
	// Convert the double matrix into binary
	convertToBinary(KERDIM, h_ker, (uint32_t) ceil(KERDIM*KERDIM/8.0), h_ker_bin);
	// TODO DEBUG: Print the double matrix.
	//printMatrix(KERDIM, h_ker);
	// TODO DEBUG: Print the binary matrix.
	if(argc == 4)
		printBinary(KERDIM, (uint32_t) ceil(KERDIM*KERDIM/8.0), h_ker_bin);

	// Copy all the matrices to the device (except the result matrices)
	hipMemcpy(d_mat, h_mat, mat_size, hipMemcpyHostToDevice);
	hipMemcpy(d_ker, h_ker, ker_size, hipMemcpyHostToDevice);
	hipMemcpy(d_mat_bin, h_mat_bin, mat_bin_size, hipMemcpyHostToDevice);
	hipMemcpy(d_ker_bin, h_ker_bin, ker_bin_size, hipMemcpyHostToDevice);

	hipMemcpy(d_MATDIM, h_MATDIM, sizeof(uint32_t), hipMemcpyHostToDevice);
	hipMemcpy(d_KERDIM, h_KERDIM, sizeof(uint32_t), hipMemcpyHostToDevice);

	// // Compute the different modes of convolution
	// clock_gettime(CLOCK_MONOTONIC, &tstart);
	// convStandard<<<grid_size, N>>>(d_MATDIM, d_KERDIM, d_mat, d_ker, d_res_standard);
	// clock_gettime(CLOCK_MONOTONIC, &tend);
	// elapsed = ((double)tend.tv_sec + 1.0e-9*tend.tv_nsec) - ((double)tstart.tv_sec + 1.0e-9*tstart.tv_nsec);
	// cout << "Standard convolution took " << elapsed << " seconds.\n";

	cout << "\n----------Binary weights----------\n";
	//Run and measure time for newConvBinW
	clock_gettime(CLOCK_MONOTONIC, &tstart);
	newConvBinW<<<grid_size, N>>>(d_MATDIM, d_KERDIM, d_mat, d_ker_bin, new_d_res_binW);
	clock_gettime(CLOCK_MONOTONIC, &tend);
	elapsed = ((double)tend.tv_sec + 1.0e-9*tend.tv_nsec) - ((double)tstart.tv_sec + 1.0e-9*tstart.tv_nsec);
	cout << "Binary weights took " << elapsed << " nanoseconds.\n";

	//Run and measure time for old convBinW
	clock_gettime(CLOCK_MONOTONIC, &tstart);
	convBinW<<<grid_size, N>>>(d_MATDIM, d_KERDIM, d_mat, d_ker_bin, d_res_binW);
	clock_gettime(CLOCK_MONOTONIC, &tend);
	elapsed = ((double)tend.tv_sec + 1.0e-9*tend.tv_nsec) - ((double)tstart.tv_sec + 1.0e-9*tstart.tv_nsec);
	cout << "Binary weights took " << elapsed << " nanoseconds.\n";

	cout << "\n----------Binary weights Binary Inputs----------\n";
	//Run and measure time for newConvBinWBinI
	clock_gettime(CLOCK_MONOTONIC, &tstart);
	newConvBinWBinI<unsigned char><<<grid_size, N>>>(d_MATDIM, d_KERDIM, d_mat_bin, d_ker_bin, new_d_res_binWbinI);
	clock_gettime(CLOCK_MONOTONIC, &tend);
	elapsed = ((double)tend.tv_sec + 1.0e-9*tend.tv_nsec) - ((double)tstart.tv_sec + 1.0e-9*tstart.tv_nsec);
	cout << "Byte wise Binary inputs and binary weights took " << elapsed << " nanoseconds.\n";
	cout << elapsed << "\n";

	//Run and measure time for old convBinWBinI
	clock_gettime(CLOCK_MONOTONIC, &tstart);
	convBinWBinI<<<grid_size, N>>>(d_MATDIM, d_KERDIM, d_mat_bin, d_ker_bin, d_res_binWbinI);
	clock_gettime(CLOCK_MONOTONIC, &tend);
	elapsed = ((double)tend.tv_sec + 1.0e-9*tend.tv_nsec) - ((double)tstart.tv_sec + 1.0e-9*tstart.tv_nsec);
	cout << "Binary inputs and binary weights took " << elapsed << " nanoseconds.\n";
	cout << elapsed << "\n";

	// Fetch the results from device
	// hipMemcpy(h_res_standard, d_res_standard, res_standard_size, hipMemcpyDeviceToHost);
	hipMemcpy(h_res_binW, d_res_binW, res_binW_size, hipMemcpyDeviceToHost);
	hipMemcpy(new_h_res_binW, new_d_res_binW, res_binW_size, hipMemcpyDeviceToHost);
	hipMemcpy(h_res_binWbinI, d_res_binWbinI, res_binWbinI_size, hipMemcpyDeviceToHost);
	hipMemcpy(new_h_res_binWbinI, new_d_res_binWbinI, res_binWbinI_size, hipMemcpyDeviceToHost);

	// TODO DEBUG: Print the results
	// cout << "Standard convolution DOUBLExDOUBLE\n";
	// printMatrix(MATDIM-KERDIM+1, h_res_standard);
	// cout << "Binary weight convolution DOUBLExBITS\n";
	cout << "\n----------Reuslt for old binary Weights----------\n";
	printMatrix(MATDIM-KERDIM+1, h_res_binW);
	cout << "\n----------Reuslt for new binary Wieghts----------\n";
	printMatrix(MATDIM-KERDIM+1, new_h_res_binW);
	if(argc == 4)
	{
		cout << "Binary weights and binary inputs BITSxBITS\n";
		cout << "dim: " << MATDIM-KERDIM+1 << "x" << MATDIM-KERDIM+1 << "\n{\n";
		for (uint32_t i = 0; i < MATDIM-KERDIM+1; i++) {
			for (uint32_t j = 0; j < MATDIM-KERDIM+1; j++) {
				cout << (uint32_t) h_res_binWbinI[i*(MATDIM-KERDIM+1)+j] << ", ";
			}
			cout << '\n';
		}
		cout << "}\n";

		cout << "NEW Binary weights and binary inputs BITSxBITS\n";
		cout << "dim: " << MATDIM-KERDIM+1 << "x" << MATDIM-KERDIM+1 << "\n{\n";
		for (uint32_t i = 0; i < MATDIM-KERDIM+1; i++) {
			for (uint32_t j = 0; j < MATDIM-KERDIM+1; j++) {
				cout << (uint32_t) new_h_res_binWbinI[i*(MATDIM-KERDIM+1)+j] << ", ";
			}
			cout << '\n';
		}
		cout << "}\n";
	}

	hipFree(d_mat);
	hipFree(d_ker);
	hipFree(d_mat_bin);
	hipFree(d_ker_bin);
	hipFree(d_res_standard);
	hipFree(d_res_binW);
	hipFree(new_d_res_binW);
	hipFree(d_res_binWbinI);
	hipFree(new_d_res_binWbinI);
	hipFree(d_MATDIM);
	hipFree(d_KERDIM);

	return 0;
}
