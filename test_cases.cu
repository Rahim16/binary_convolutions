#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <iostream>
#include <cstdlib>
#include "convolution.h"
#include "array_management.h"

void convBinWBinIWrapper(uint32_t blocks, uint32_t threads, uint32_t MATDIM, uint32_t KERDIM, unsigned char* matrix, unsigned char* kernel, unsigned char* result, bool old)
{
  //Calculate Array size
  uint32_t res_size = (MATDIM-KERDIM+1)*(MATDIM-KERDIM+1) * sizeof(unsigned char);
  uint32_t mat_size = (uint32_t) ceil(MATDIM*MATDIM/8.0) * sizeof(unsigned char);
	uint32_t ker_size = (uint32_t) ceil(KERDIM*KERDIM/8.0) * sizeof(unsigned char);

  //Mallocate device data
  uint32_t *d_MATDIM, *d_KERDIM;
  hipMalloc((void**) &d_MATDIM, sizeof(uint32_t));
  hipMalloc((void**) &d_KERDIM, sizeof(uint32_t));

  unsigned char *d_mat, *d_ker, *d_res;
  hipMalloc((void**) &d_mat, mat_size);
	hipMalloc((void**) &d_ker, ker_size);
  hipMalloc((void**) &d_res, res_size);

  //Copy input to device
  hipMemcpy(d_MATDIM, &MATDIM, sizeof(uint32_t), hipMemcpyHostToDevice);
  hipMemcpy(d_KERDIM, &KERDIM, sizeof(uint32_t), hipMemcpyHostToDevice);

  hipMemcpy(d_ker, kernel, ker_size, hipMemcpyHostToDevice);
  hipMemcpy(d_mat, matrix, mat_size, hipMemcpyHostToDevice);

  if(old)
    convBinWBinI<<<blocks, threads>>>(d_MATDIM, d_KERDIM, d_mat, d_ker, d_res);
  else
    newConvBinWBinI<<<blocks, threads>>>(d_MATDIM, d_KERDIM, d_mat, d_ker, d_res);

  //copyResult back to host
  hipMemcpy(result, d_res, res_size, hipMemcpyDeviceToHost);

  hipFree(d_MATDIM);
  hipFree(d_KERDIM);
  hipFree(d_mat);
  hipFree(d_ker);
  hipFree(d_res);
}



TEST(binIbinW_Test, OneElement)
{
  //Input Data
  uint32_t MATDIM = 1;
  uint32_t KERDIM = 1;
  unsigned char matrix[1] = {0b10000000};
  unsigned char kernel[1] = {0b10000000};
  unsigned char result[1];

  convBinWBinIWrapper(1, 1, MATDIM, KERDIM, matrix, kernel, result, false);

  ASSERT_EQ(result[0], 1);
}

TEST(binIbinW_Test, 5Matrix3Kernel)
{
  uint32_t MATDIM = 5;
  uint32_t KERDIM = 3;
  /*  Matrix        Kernel
      11010        101
      10001        011
      00110        001
      11111
      00000
    */
  unsigned char matrix[4] = { 0b11010100, 0b01001101, 0b11110000, 0b00000000 };
  unsigned char kernel[2] = {0b10101100, 0b10000000};
  unsigned char result[9];

  convBinWBinIWrapper(1, 9, MATDIM, KERDIM, matrix, kernel, result, false);

  unsigned char suspacted_result[9] = {4, 6, 2, 5, 5,  4, 6, 5, 5};

  for(int i = 0; i < 9; i++)
      ASSERT_EQ(result[i], suspacted_result[i]);
}

TEST(binIBinW_Test, OldAsOracle)
{
  uint32_t MATDIM = ceil( (rand()/RAND_MAX) * 1024);
  uint32_t KERDIM = 1;
  while(KERDIM%2)
    KERDIM = ceil( (rand()/RAND_MAX) * MATDIM);

  std::cerr << "[          ]" << MATDIM << " " << KERDIM << "\n";

  double* double_matrix = new double[ MATDIM * MATDIM * sizeof(double)];
  double* double_kernel = new double[ KERDIM * KERDIM * sizeof(double)];

  initMat(MATDIM, double_matrix);
  initMat(KERDIM, double_kernel);

  unsigned char* bin_matrix = new unsigned char[ (uint32_t)ceil(MATDIM*MATDIM/8.0) ];
  unsigned char* bin_kernel = new unsigned char[ (uint32_t)ceil(KERDIM*KERDIM/8.0) ];
  unsigned char* result = new unsigned char[ (MATDIM-KERDIM+1)*(MATDIM-KERDIM+1) ];
  unsigned char* suspected_result = new unsigned char[ (MATDIM-KERDIM+1)*(MATDIM-KERDIM+1) ];

  convertToBinary(MATDIM, double_matrix, (uint32_t)ceil(MATDIM*MATDIM/8.0), bin_matrix);
  convertToBinary(KERDIM, double_kernel, (uint32_t)ceil(KERDIM*KERDIM/8.0), bin_kernel);

  uint32_t N = 10;
  uint32_t grid_size = ceil((MATDIM-KERDIM+1) * (MATDIM-KERDIM+1) / ((double) N));

  convBinWBinIWrapper(grid_size, N, MATDIM, KERDIM, bin_matrix, bin_kernel, suspected_result, true);
  convBinWBinIWrapper(grid_size, N, MATDIM, KERDIM, bin_matrix, bin_kernel, result, false);

  for(int i=0; i < (MATDIM-KERDIM+1)*(MATDIM-KERDIM+1); i++)
    ASSERT_EQ(result[i], suspected_result[i]);
}


int main(int argc, char **argv)
{
  testing::InitGoogleTest(&argc, argv);
  return RUN_ALL_TESTS();
}
